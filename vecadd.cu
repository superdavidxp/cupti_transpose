/*
 * Copyright 2011-2015 NVIDIA Corporation. All rights reserved
 *
 * Sample app to demonstrate use of CUPTI library to obtain metric values
 * using callbacks for CUDA runtime APIs
 *
 */

#include <stdio.h>
#include <hip/hip_runtime.h>

#define DRIVER_API_CALL(apiFuncCall)                                           \
do {                                                                           \
    hipError_t _status = apiFuncCall;                                            \
    if (_status != hipSuccess) {                                             \
        fprintf(stderr, "%s:%d: error: function %s failed with error %d.\n",   \
                __FILE__, __LINE__, #apiFuncCall, _status);                    \
        exit(-1);                                                              \
    }                                                                          \
} while (0)

#define RUNTIME_API_CALL(apiFuncCall)                                          \
do {                                                                           \
    hipError_t _status = apiFuncCall;                                         \
    if (_status != hipSuccess) {                                              \
        fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n",   \
                __FILE__, __LINE__, #apiFuncCall, hipGetErrorString(_status));\
        exit(-1);                                                              \
    }                                                                          \
} while (0)

#define TILE_DIM 64
#define BLOCK_ROWS 8
#define NUM_REPS 100

// Device code
__global__ void VecAdd(const float* A, const float* B, float* C, int N)
{
  __shared__ float tile[TILE_DIM];
  tile[threadIdx.x] = A[threadIdx.x];

  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < N)
    C[i] = A[i] + B[i];
}

static void
initVec(float *vec, int n)
{
  for (int i=0; i< n; i++)
    vec[i] = i;
}

static void
cleanUp(float *h_A, float *h_B, float *h_C, float *d_A, float *d_B, float *d_C)
{
  if (d_A)
    hipFree(d_A);
  if (d_B)
    hipFree(d_B);
  if (d_C)
    hipFree(d_C);

  // Free host memory
  if (h_A)
    free(h_A);
  if (h_B)
    free(h_B);
  if (h_C)
    free(h_C);
}

static void
runPass()
{
  int N = 10240 * 10240;
  size_t size = N * sizeof(float);
  int threadsPerBlock = 0;
  int blocksPerGrid = 0;
  float *h_A, *h_B, *h_C;
  float *d_A, *d_B, *d_C;
  int i, sum;

  // Allocate input vectors h_A and h_B in host memory
  h_A = (float*)malloc(size);
  h_B = (float*)malloc(size);
  h_C = (float*)malloc(size);

  // Initialize input vectors
  initVec(h_A, N);
  initVec(h_B, N);
  memset(h_C, 0, size);

  // Allocate vectors in device memory
  hipMalloc((void**)&d_A, size);
  hipMalloc((void**)&d_B, size);
  hipMalloc((void**)&d_C, size);

  // Copy vectors from host memory to device memory
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

  hipMemset(d_A, 0, size);
  hipMemset(d_B, 0, size);

  // Invoke kernel
  threadsPerBlock = 64;
  blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
  printf("Launching kernel: blocks %d, thread/block %d\n",
         blocksPerGrid, threadsPerBlock);
  VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

  // Copy result from device memory to host memory
  // h_C contains the result in host memory
  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

  cleanUp(h_A, h_B, h_C, d_A, d_B, d_C);
}

int
main(int argc, char *argv[])
{

  runPass();

  return 0;
}
