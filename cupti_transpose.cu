#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <cupti.h>

#define CUPTI_CALL(call)                                                    \
do {                                                                        \
    CUptiResult _status = call;                                             \
    if (_status != CUPTI_SUCCESS) {                                         \
      const char *errstr;                                                   \
      cuptiGetResultString(_status, &errstr);                               \
      fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n",  \
              __FILE__, __LINE__, #call, errstr);                           \
      exit(-1);                                                             \
    }                                                                       \
} while (0)

#define DRIVER_API_CALL(apiFuncCall)                                           \
do {                                                                           \
    hipError_t _status = apiFuncCall;                                            \
    if (_status != hipSuccess) {                                             \
        fprintf(stderr, "%s:%d: error: function %s failed with error %d.\n",   \
                __FILE__, __LINE__, #apiFuncCall, _status);                    \
        exit(-1);                                                              \
    }                                                                          \
} while (0)

#define RUNTIME_API_CALL(apiFuncCall)                                          \
do {                                                                           \
    hipError_t _status = apiFuncCall;                                         \
    if (_status != hipSuccess) {                                              \
        fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n",   \
                __FILE__, __LINE__, #apiFuncCall, hipGetErrorString(_status));\
        exit(-1);                                                              \
    }                                                                          \
} while (0)

#define BUF_SIZE (8 * 1024)
#define ALIGN_SIZE (8)
#define ALIGN_BUFFER(buffer, align)                                            \
    (((uintptr_t) (buffer) & ((align)-1)) ? ((buffer) + (align) - ((uintptr_t) (buffer) & ((align)-1))) : (buffer))

#define TILE_DIM 32
#define BLOCK_ROWS 8
#define NUM_REPS 100

static const char *
getUvmCounterKindString(CUpti_ActivityUnifiedMemoryCounterKind kind)
{
    switch (kind)
    {
    case CUPTI_ACTIVITY_UNIFIED_MEMORY_COUNTER_KIND_BYTES_TRANSFER_HTOD:
        return "BYTES_TRANSFER_HTOD";
    case CUPTI_ACTIVITY_UNIFIED_MEMORY_COUNTER_KIND_BYTES_TRANSFER_DTOH:
        return "BYTES_TRANSFER_DTOH";
    default:
        break;
    }
    return "<unknown>";
}

static void
printActivity(CUpti_Activity *record)
{
    switch (record->kind)
    {
    case CUPTI_ACTIVITY_KIND_UNIFIED_MEMORY_COUNTER:
        {
            CUpti_ActivityUnifiedMemoryCounter2 *uvm = (CUpti_ActivityUnifiedMemoryCounter2 *)record;
            printf("UNIFIED_MEMORY_COUNTER [ %llu %llu ] kind=%s value=%llu src %u dst %u\n",
                (unsigned long long)(uvm->start),
                (unsigned long long)(uvm->end),
                getUvmCounterKindString(uvm->counterKind),
                (unsigned long long)uvm->value,
                uvm->srcId,
                uvm->dstId);
            break;
        }
    case CUPTI_ACTIVITY_KIND_MEMCPY:
        {
            CUpti_ActivityMemcpy *uvm = (CUpti_ActivityMemcpy *) record;
            printf( "MEMORY_CPY [ ID %d/%d/%d :: %llu (ms) :: %llu (bytes) :: %d/%d/%d (Kind)]\n",
                    (int) uvm->contextId, (int) uvm->correlationId, (int) uvm->deviceId,
                    (unsigned long long) ((uvm->end-uvm->start)/1e6),
                    (unsigned long long) (uvm->bytes),
                    uvm->copyKind, uvm->srcKind, uvm->dstKind);
            break;
        }
    case CUPTI_ACTIVITY_KIND_MEMSET:
        {
            CUpti_ActivityMemset *uvm = (CUpti_ActivityMemset *) record;
            printf( "MEMORY_SET [ ID %d/%d/%d :: %llu (ms) :: %llu (bytes) :: %d (Kind) :: %d (value) ]\n",
                    (int) uvm->contextId, (int) uvm->correlationId, (int) uvm->deviceId,
                    (unsigned long long) ((uvm->end-uvm->start)/1e6),
                    (unsigned long long) (uvm->bytes),
                    uvm->memoryKind,
                    uvm->value);
            break;
        }
    case CUPTI_ACTIVITY_KIND_KERNEL:
    case CUPTI_ACTIVITY_KIND_CONCURRENT_KERNEL:
        {
            CUpti_ActivityKernel3 *uvm = (CUpti_ActivityKernel3 *) record;
            printf( "KERNEL_RUN [ ID %d/%d/%d :: %llu (ms) :: %d/%d/%d (block) :: %d (gridID) :: \
%d (RegPerThd) :: %d/%d (SharedMem) :: %d/%d (LocMem) ]\n",
                    (int) uvm->contextId, (int) uvm->correlationId, (int) uvm->deviceId,
                    (unsigned long long) ((uvm->end-uvm->start)/1e6),
                    (int) uvm->blockX, (int) uvm->blockY, (int) uvm->blockZ,
                    (int) uvm->gridId, (int) uvm->registersPerThread,
                    (int) uvm->dynamicSharedMemory, (int) uvm->staticSharedMemory,
                    (int) uvm->localMemoryPerThread, (int) uvm->localMemoryTotal
                  );
            break;
        }
    default:
        printf("  <unknown>\n");
        break;
    }
}

static void CUPTIAPI
bufferRequested(uint8_t **buffer, size_t *size, size_t *maxNumRecords)
{
    uint8_t *rawBuffer;

    *size = BUF_SIZE;
    rawBuffer = (uint8_t *)malloc(*size + ALIGN_SIZE);

    *buffer = ALIGN_BUFFER(rawBuffer, ALIGN_SIZE);
    *maxNumRecords = 0;

    if (*buffer == NULL) {
        printf("Error: out of memory\n");
        exit(-1);
    }
}

static void CUPTIAPI
bufferCompleted(hipCtx_t ctx, uint32_t streamId, uint8_t *buffer, size_t size, size_t validSize)
{
    CUptiResult status;
    CUpti_Activity *record = NULL;

    do {
        status = cuptiActivityGetNextRecord(buffer, validSize, &record);
        if (status == CUPTI_SUCCESS) {
            printActivity(record);
        }
        else if (status == CUPTI_ERROR_MAX_LIMIT_REACHED) {
            break;
        }
        else {
            CUPTI_CALL(status);
        }
    } while (1);

    // report any records dropped from the queue
    size_t dropped;
    CUPTI_CALL(cuptiActivityGetNumDroppedRecords(ctx, streamId, &dropped));
    if (dropped != 0) {
        printf("Dropped %u activity records\n", (unsigned int)dropped);
    }

    free(buffer);
}

template<class T>
__host__ __device__ void checkData(const char *loc, T *data, int size, int expectedVal) {
    int i;

    for (i = 0; i < size / (int)sizeof(T); i++) {
        if (data[i] != expectedVal) {
            printf("Mismatch found on %s\n", loc);
            printf("Address 0x%p, Observed = 0x%x Expected = 0x%x\n", data+i, data[i], expectedVal);
            break;
        }
    }
}

template<class T>
__host__ __device__ void writeData(T *data, int size, int writeVal) {
    int i;

    for (i = 0; i < size / (int)sizeof(T); i++) {
        data[i] = writeVal;
    }
}

// No bank-conflict transpose
// Same as transposeCoalesced except the first tile dimension is padded
// to avoid shared memory bank conflicts.
__global__ void transposeNoBankConflicts(float *odata, const float *idata)
{
  __shared__ float tile[TILE_DIM][TILE_DIM+1];

  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*width + x];

  __syncthreads();

  x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
  y = blockIdx.x * TILE_DIM + threadIdx.y;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     odata[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
}

int main(int argc, char **argv)
{
    const int N = 1024*10;
    const int mem_size = N*N*sizeof(float);

    int deviceCount;
    float* h_idata = NULL;
    float* h_tdata = NULL;
    float* d_idata = NULL;
    float* d_tdata = NULL;

    dim3 dimGrid(N/TILE_DIM, N/TILE_DIM, 1);
    dim3 dimBlock(TILE_DIM, BLOCK_ROWS, 1);

    DRIVER_API_CALL( hipInit(0) );

    DRIVER_API_CALL( hipGetDeviceCount(&deviceCount) );

    if (deviceCount == 0) {
        printf("|    there is no device supporting CUDA.\n");
        exit(-1);
    }

    // register cupti activity buffer callbacks
    CUPTI_CALL( cuptiActivityRegisterCallbacks(bufferRequested, bufferCompleted) );

    CUPTI_CALL( cuptiActivityEnable(CUPTI_ACTIVITY_KIND_MEMCPY) );
    CUPTI_CALL( cuptiActivityEnable(CUPTI_ACTIVITY_KIND_MEMSET) );
    CUPTI_CALL( cuptiActivityEnable(CUPTI_ACTIVITY_KIND_KERNEL) );

    // allocate memory
    printf("|    allocation size in %d bytes\n", 2*mem_size);
    RUNTIME_API_CALL( hipHostMalloc((void**)&h_idata, mem_size, hipHostMallocDefault) );
    RUNTIME_API_CALL( hipHostMalloc((void**)&h_tdata, mem_size, hipHostMallocDefault) );

    // initialize memory
    RUNTIME_API_CALL( hipMalloc((void**)&d_idata, mem_size) );
    RUNTIME_API_CALL( hipMalloc((void**)&d_tdata, mem_size) );

    // transfer memory
    RUNTIME_API_CALL( hipMemcpy(d_idata, h_idata, mem_size, hipMemcpyHostToDevice) );

    // memory setting
    RUNTIME_API_CALL( hipMemset(d_idata, 100, mem_size) );

    // ------------------------
    // transposeNoBankConflicts
    // ------------------------
    printf("|    conflict-free transpose\n");
    RUNTIME_API_CALL( hipMemset(d_tdata, 0, mem_size) );

    // events for timing
    hipEvent_t startEvent, stopEvent;
    RUNTIME_API_CALL( hipEventCreate(&startEvent) );
    RUNTIME_API_CALL( hipEventCreate(&stopEvent) );
    float ms;

    // warmup
    RUNTIME_API_CALL( hipEventRecord(startEvent, 0) );
    transposeNoBankConflicts<<<dimGrid, dimBlock>>>(d_tdata, d_idata);
    // for (int i = 0; i < NUM_REPS; i++)
    //     transposeNoBankConflicts<<<dimGrid, dimBlock>>>(d_tdata, d_idata);
    RUNTIME_API_CALL( hipEventRecord(stopEvent, 0) );
    RUNTIME_API_CALL( hipEventSynchronize(stopEvent) );
    RUNTIME_API_CALL( hipEventElapsedTime(&ms, startEvent, stopEvent) );
    RUNTIME_API_CALL( hipMemcpy(h_tdata, d_tdata, mem_size, hipMemcpyDeviceToHost) );
    // postprocess(gold, h_tdata, nx * ny, ms);

    RUNTIME_API_CALL( hipDeviceSynchronize() );

    // free host memory
    RUNTIME_API_CALL(hipHostFree(h_idata));
    RUNTIME_API_CALL(hipHostFree(h_tdata));

    // free device memory
    RUNTIME_API_CALL(hipFree(d_idata));
    RUNTIME_API_CALL(hipFree(d_tdata));

    CUPTI_CALL(cuptiActivityFlushAll(0));

    // disable unified memory counter activity
    CUPTI_CALL( cuptiActivityDisable(CUPTI_ACTIVITY_KIND_MEMCPY) );
    CUPTI_CALL( cuptiActivityDisable(CUPTI_ACTIVITY_KIND_MEMSET) );
    CUPTI_CALL( cuptiActivityDisable(CUPTI_ACTIVITY_KIND_KERNEL) );

    printf("|    time %f\n", ms);

    hipDeviceReset();

    return 0;
}
